
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


__global__ void VecAdd(float *color, unsigned int atoms)
 {
     int j = threadIdx.x +blockDim.x *blockIdx.x;
        
	if(j < atoms) //if index is less than 104014
          {
         	 if(color[j] < 0.45)  //if Array is less 45%
                  {
                  	color[j] = .000001; //then the glass is all shattered 
                  }
           } 
}

